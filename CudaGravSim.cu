#include "hip/hip_runtime.h"
#include <cmath>

#include "CudaGravSim.h"

__global__ void getAllForces(CudaGravSim::Particle* particles,double* forces, double g, uint32_t size)
{
    uint32_t particleId = blockDim.x * blockIdx.x + threadIdx.x;
    if(particleId>=size)
    {
        return;
    }
    forces[particleId*3+0] = 0;
    forces[particleId*3+1] = 0;
    forces[particleId*3+2] = 0;
    for(uint32_t index = 0; index<size; index++)
    {
        if (particleId != index)
        {

            double deltaX = particles[particleId].pos.x - particles[index].pos.x;
            double deltaY = particles[particleId].pos.y - particles[index].pos.y;
            double deltaZ = particles[particleId].pos.z - particles[index].pos.z;

            double absoluteDistance = sqrt(pow(deltaX, 2) + pow(deltaY, 2) + pow(deltaZ, 2));
            double totalForce = g * (particles[particleId].mass * particles[index].mass) /pow(absoluteDistance,3);

            forces[index*3+0] += totalForce*deltaX;
            forces[index*3+1] += totalForce*deltaY;
            forces[index*3+2] += totalForce*deltaZ;
        }
    }
}

__global__ void getFinalPosition(double* forces, CudaGravSim::Particle* particles, uint32_t size, double timeStep)
{
    uint32_t particleId = blockDim.x * blockIdx.x + threadIdx.x;
    if(particleId>=size)
    {
        return;
    }
    double time = pow(timeStep,2);
    particles[particleId].pos.x += particles[particleId].vel.x*timeStep+(forces[particleId*3+0]/particles[particleId].mass*time)/2.0;
    particles[particleId].pos.y += particles[particleId].vel.y*timeStep+(forces[particleId*3+1]/particles[particleId].mass*time)/2.0;
    particles[particleId].pos.z += particles[particleId].vel.z*timeStep+(forces[particleId*3+2]/particles[particleId].mass*time)/2.0;
}

__global__ void getFinalVelocity(double* forces, CudaGravSim::Particle* particles, uint32_t size, double timeStep)
{
    uint32_t particleId = blockDim.x * blockIdx.x + threadIdx.x;
    if(particleId>=size)
    {
        return;
    }
    particles[particleId].vel.x += forces[particleId*3+0]/particles[particleId].mass*timeStep;
    particles[particleId].vel.y += forces[particleId*3+1]/particles[particleId].mass*timeStep;
    particles[particleId].vel.z += forces[particleId*3+2]/particles[particleId].mass*timeStep;
}

void CudaGravSim::initArrays(std::vector<VulkanApp::Vertex> vertecies)
{
    size = vertecies.size();
    hipMalloc(&particleArrayDevice,sizeof(Particle)*size);
    hipMalloc(&forces,sizeof(double)*size*3);
    particleArrayHost = static_cast<Particle *>(malloc(sizeof(Particle) * size));
    for(uint32_t i = 0; i<size; i++)
    {
        particleArrayHost[i].pos = vertecies[i].pos;
        particleArrayHost[i].mass = 10;
        particleArrayHost[i].vel = {0,0,0};
    }
    hipMemcpy(particleArrayDevice,particleArrayHost,size*sizeof(Particle),hipMemcpyHostToDevice);
}

void CudaGravSim::cleanup()
{
    free(particleArrayHost);
    hipFree(particleArrayDevice);
    hipFree(forces);
}

void CudaGravSim::step()
{
    getAllForces<<<std::ceil(size/1024.0),1024>>>(particleArrayDevice,forces,gravitationalConstant,size);
    getFinalPosition<<<std::ceil(size/1024.0),1024>>>(forces, particleArrayDevice,size,timeStep);
    getFinalVelocity<<<std::ceil(size/1024.0),1024>>>(forces,particleArrayDevice,size,timeStep);
    sync();
}

void CudaGravSim::copyData(std::vector<VulkanApp::Vertex> *vertecies)
{
    std::vector<VulkanApp::Vertex>& vert = *vertecies;
    for(uint32_t i = 0; i<size; i++)
    {
        vert[i].pos = particleArrayHost[i].pos;
    }
}

void CudaGravSim::sync()
{
    double* temp;
    temp = (double*) malloc(sizeof(double )*3*size);
    hipMemcpy(particleArrayHost,particleArrayDevice,size*sizeof(Particle),hipMemcpyDeviceToHost);
    hipMemcpy(temp,forces,size*sizeof(double)*3,hipMemcpyDeviceToHost);
    free(temp);
}